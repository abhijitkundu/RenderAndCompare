#include "hip/hip_runtime.h"
/**
 * @file SegmentationAccuracy.cu
 * @brief SegmentationAccuracy
 *
 * @author Abhijit Kundu
 */

#include "SegmentationAccuracy.h"

namespace RaC {

float computeIoUwithCUDA(const Eigen::Tensor<uint8_t, 4, Eigen::RowMajor>& gt_images,
                         const Eigen::Tensor<uint8_t, 4, Eigen::RowMajor>& pred_images) {
  if (gt_images.size() != pred_images.size())
     throw std::runtime_error("Dimension mismatch: gt_images.dimensions() ! = pred_images.dimensions()");
  const std::size_t gt_images_bytes = gt_images.size()  * sizeof(uint8_t);
  const std::size_t pred_images_bytes = pred_images.size()  * sizeof(uint8_t);

  uint8_t* d_gt_images;
  uint8_t* d_pred_images;

  cudaCheckError(hipMalloc((void**)(&d_gt_images), gt_images_bytes));
  cudaCheckError(hipMalloc((void**)(&d_pred_images), pred_images_bytes));

  cudaCheckError(hipMemcpy(d_gt_images, gt_images.data(), gt_images_bytes, hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_pred_images, pred_images.data(), pred_images_bytes, hipMemcpyHostToDevice));

  GpuTimer gpu_timer;
  gpu_timer.Start();

  float mean_iou = 0;


  gpu_timer.Stop();
  float elapsed_millis = gpu_timer.ElapsedMillis();
  std::cout << "GPU Time = " << elapsed_millis << " ms\n";

  cudaCheckError(hipFree((void*)d_gt_images));
  cudaCheckError(hipFree((void*)d_pred_images));

  return mean_iou;
}

template <typename ImageScalar, typename HistScalar>
__global__ void histogram_atomics(const ImageScalar* const image, int width, int height, HistScalar *hist) {
  // pixel coordinates
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // grid dimensions
  int nx = blockDim.x * gridDim.x;
  int ny = blockDim.y * gridDim.y;

  for (int col = x; col < width; col += nx) {
    for (int row = y; row < height; row += ny) {
      int label = static_cast<int>(image[row * width + col]);
      atomicAdd(&hist[label] , 1 );
    }
  }
}

void computeHistogramWithAtomics(const uint8_t* const image, int width, int height, int *hist, int num_labels) {
  using ImageScalar = uint8_t;
  using HistScalar = int;

  ImageScalar* d_image;
  const std::size_t image_bytes = width * height * sizeof(ImageScalar);
  cudaCheckError(hipMalloc(&d_image, image_bytes));
  cudaCheckError(hipMemcpy(d_image, image, image_bytes, hipMemcpyHostToDevice));

  HistScalar *d_hist;
  cudaCheckError(hipMalloc(&d_hist, num_labels * sizeof(HistScalar)));


  GpuTimer gpu_timer;
  gpu_timer.Start();

  cudaCheckError(hipMemset(d_hist, 0, num_labels * sizeof(HistScalar)));

  dim3 block(16, 16);
  dim3 grid((width + 16 - 1) / 16 , (height + 16 - 1) / 16 ) ;

  histogram_atomics<<<grid, block>>>(d_image, width, height, d_hist);

  gpu_timer.Stop();
  std::cout << "GPU Time = " << gpu_timer.ElapsedMillis() << " ms\n";

  cudaCheckError(hipMemcpy(hist, d_hist, num_labels * sizeof(HistScalar), hipMemcpyDeviceToHost));

  cudaCheckError(hipFree((void*)d_image));
  cudaCheckError(hipFree((void*)d_hist));
}

template <int NumBins, int NumParts, typename ImageScalar, typename HistScalar>
__global__ void histogram_smem_atomics(const ImageScalar* const image, int width, int height, HistScalar *out) {
  // pixel coordinates
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // grid dimensions
  int nx = blockDim.x * gridDim.x;
  int ny = blockDim.y * gridDim.y;


  // linear thread index within 2D block
  int t = threadIdx.x + threadIdx.y * blockDim.x;

  // total threads in 2D block
  int nt = blockDim.x * blockDim.y;

  // linear block index within 2D grid
  int g = blockIdx.x + blockIdx.y * gridDim.x;

  // initialize temporary hist array in shared memory
  __shared__ HistScalar smem[NumBins + 1];
  for (int i = t; i < NumBins + 1; i += nt)
    smem[i] = 0;
  __syncthreads();

  for (int col = x; col < width; col += nx) {
    for (int row = y; row < height; row += ny) {
      int label = static_cast<int>(image[row * width + col]);
      atomicAdd(&smem[label] , 1 );
    }
  }
  __syncthreads();

  // write partial histogram into the global memory
  out += g * NumParts;
  for (int i = t; i < NumBins; i += nt) {
    out[i] = smem[i];
  }
}

template <int NumBins, int NumParts, typename HistScalar>
__global__ void histogram_smem_accum(const HistScalar *in, int n, HistScalar *out)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < NumBins) {
    HistScalar total = 0;
    for (int j = 0; j < n; j++)
      total += in[i + NumParts * j];
    out[i] = total;
  }
}

void computeHistogramWithSharedAtomics(const uint8_t* const image, int width, int height, int *hist, int num_labels) {
  using ImageScalar = uint8_t;
  using HistScalar = int;

  ImageScalar* d_image;
  const std::size_t image_bytes = width * height * sizeof(ImageScalar);
  cudaCheckError(hipMalloc(&d_image, image_bytes));
  cudaCheckError(hipMemcpy(d_image, image, image_bytes, hipMemcpyHostToDevice));

  HistScalar *d_hist;
  hipMalloc(&d_hist, num_labels * sizeof(HistScalar));

  static const int NumBins = 25;
  static const int NumParts = 1024;

  dim3 block(32, 4);
  dim3 grid(16, 16);
  int total_blocks = grid.x * grid.y;

  // allocate partial histogram
  HistScalar *d_part_hist;
  hipMalloc(&d_part_hist, total_blocks * NumParts * sizeof(HistScalar));

  dim3 block2(128);
  dim3 grid2((NumBins + block.x - 1) / block.x);

  GpuTimer gpu_timer;
  gpu_timer.Start();

  histogram_smem_atomics<NumBins, NumParts><<<grid, block>>>(d_image, width, height, d_part_hist);

  histogram_smem_accum<NumBins, NumParts><<<grid2, block2>>>(d_part_hist, total_blocks, d_hist);

  gpu_timer.Stop();
  std::cout << "GPU Time = " << gpu_timer.ElapsedMillis() << " ms\n";

  cudaCheckError(hipMemcpy(hist, d_hist, NumBins * sizeof(HistScalar), hipMemcpyDeviceToHost));

  cudaCheckError(hipFree((void*)d_image));
  cudaCheckError(hipFree((void*)d_hist));
}



template <int NumBins, typename ImageScalar, typename HistScalar>
__global__ void histogram_shared_bins(const ImageScalar* const image, std::size_t size, HistScalar *hist) {

  // Initialize shared mem
  __shared__ HistScalar smem[NumBins];
  smem[threadIdx.x] = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  while (i < size) {
    int label = static_cast<int>(image[i]);
    atomicAdd(&smem[label] , 1 );
    i += offset;
  }
  __syncthreads();
  atomicAdd(&(hist[threadIdx.x]), smem[threadIdx.x]);
}


void computeHistogramWithSharedBins(const uint8_t* const image, int width, int height, int *hist, int num_labels) {
  using ImageScalar = uint8_t;
  using HistScalar = int;

  if (num_labels != 25)
    throw std::runtime_error("Only support 25 labels");

  ImageScalar* d_image;
  const std::size_t image_bytes = width * height * sizeof(ImageScalar);
  cudaCheckError(hipMalloc(&d_image, image_bytes));
  cudaCheckError(hipMemcpy(d_image, image, image_bytes, hipMemcpyHostToDevice));


  int numSMs;
  cudaCheckError(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));

  HistScalar *d_hist;
  cudaCheckError(hipMalloc(&d_hist, num_labels * sizeof(HistScalar)));

  GpuTimer gpu_timer;
  gpu_timer.Start();

  cudaCheckError(hipMemset(d_hist, 0, num_labels * sizeof(HistScalar)));

  static const int NumBins = 25;

  histogram_shared_bins<NumBins><<<numSMs*8, NumBins>>>(d_image, width * height, d_hist);

  gpu_timer.Stop();
  std::cout << "GPU Time = " << gpu_timer.ElapsedMillis() << " ms\n";

  cudaCheckError(hipMemcpy(hist, d_hist, num_labels * sizeof(HistScalar), hipMemcpyDeviceToHost));

  cudaCheckError(hipFree((void*)d_image));
  cudaCheckError(hipFree((void*)d_hist));
}

template <int NumBins, typename ImageScalar, typename HistScalar>
__global__ void histogram_private_bins(const ImageScalar* const image, std::size_t size, HistScalar *hist) {

  // Initialize private mem
  HistScalar smem[NumBins];
#pragma unroll
  for (int i = 0; i < NumBins; ++i)
    smem[i] = 0;

  const int offset = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x ;i < size; i += offset) {
    int label = static_cast<int>(image[i]);
    ++smem[label];
  }

#pragma unroll
  for (int i = 0; i < NumBins; ++i)
    atomicAdd(&(hist[i]), smem[i]);
}

void computeHistogramWithPrivateBins(const uint8_t* const image, int width, int height, int *hist, int num_labels) {
  using ImageScalar = uint8_t;
  using HistScalar = int;

  if (num_labels != 25)
    throw std::runtime_error("Only support 25 labels");

  ImageScalar* d_image;
  const std::size_t image_bytes = width * height * sizeof(ImageScalar);
  cudaCheckError(hipMalloc(&d_image, image_bytes));
  cudaCheckError(hipMemcpy(d_image, image, image_bytes, hipMemcpyHostToDevice));


  int numSMs;
  cudaCheckError(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));

  HistScalar *d_hist;
  cudaCheckError(hipMalloc(&d_hist, num_labels * sizeof(HistScalar)));

  GpuTimer gpu_timer;
  gpu_timer.Start();

  cudaCheckError(hipMemset(d_hist, 0, num_labels * sizeof(HistScalar)));
  static const int NumBins = 25;

  histogram_private_bins<NumBins><<<numSMs, 256>>>(d_image, width * height, d_hist);

  gpu_timer.Stop();
  std::cout << "GPU Time = " << gpu_timer.ElapsedMillis() << " ms\n";

  cudaCheckError(hipMemcpy(hist, d_hist, num_labels * sizeof(HistScalar), hipMemcpyDeviceToHost));

  cudaCheckError(hipFree((void*)d_image));
  cudaCheckError(hipFree((void*)d_hist));
}

}  // namespace RaC



