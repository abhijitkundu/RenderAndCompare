#include "hip/hip_runtime.h"
/**
 * @file NumericDiff.cu
 * @brief NumericDiff
 *
 * @author Abhijit Kundu
 */

#include "NumericDiff.h"
#include "CuteGL/Utils/CudaUtils.h"

namespace RaC {

template<class T>
__global__ void central_diff_kernel(const int n,
                                    const T* const fplus,
                                    const T* const fminus,
                                    const T step_size,
                                    T* diff,
                                    const int diff_stride) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    diff[i*diff_stride] = (fplus[i] - fminus[i]) / (2 * step_size);
  }
}

template <class T>
void central_diff_gpu(const int n, const T* const fplus, const T* const fminus, const T step_size, T* diff, const int diff_stride) {
  central_diff_kernel<<< (n + 32 - 1) / 32, 32>>>(n, fplus, fminus, step_size, diff, diff_stride);
}

// explicit instantiation
template void central_diff_gpu<float>(const int n,
                                      const float* const fplus,
                                      const float* const fminus,
                                      const float step_size,
                                      float* diff,
                                      const int diff_stride);
// explicit instantiation
template void central_diff_gpu<double>(const int n,
                                       const double* const fplus,
                                       const double* const fminus,
                                       const double step_size,
                                       double* diff,
                                       const int diff_stride);

}  // namespace RaC
